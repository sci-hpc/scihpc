#include <stdio.h>
#include <hip/hip_runtime.h>

void printDeviceInfo(hipDeviceProp_t prop) {

   printf("Name                         - %s\n",  prop.name);
   printf("Total global memory          - %lu MB \n", prop.totalGlobalMem/(1024*1024));
   printf("Total constant memory        - %lu KB \n", prop.totalConstMem/1024);

   printf("Shared memory per block      - %lu KB \n", prop.sharedMemPerBlock/1024);
   printf("Total registers per block    - %d\n", prop.regsPerBlock);
   printf("Maximum threads per block    - %d\n", prop.maxThreadsPerBlock);

   printf("Clock rate                   - %d\n",  prop.clockRate);
   printf("Number of multi-processors   - %d\n",  prop.multiProcessorCount);

  }

int main( ) {

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Available CUDA devices - %d\n", deviceCount);
    for (int i=0;i<deviceCount;i++){

        // Device informatioon
        printf("\nCUDA Device #%d\n", i);
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printDeviceInfo(prop);

    }
}